#include "hip/hip_runtime.h"
#include <stdio.h>
#include <iostream>
#include <hip/hip_runtime_api.h>
#include <Eigen/Eigen>

inline
hipError_t checkCuda(hipError_t result)
{
  if (result != hipSuccess) 
  {
    fprintf(stderr, "CUDA Runtime Error: %s\n", hipGetErrorString(result));
    assert(result == hipSuccess);
  }
  return result;
}

static inline int divUp(int total, int grain) { return (total + grain - 1) / grain; }

void set_device(int device_id)
{
  int current_device;
  checkCuda(hipGetDevice(&current_device));
  if (current_device == device_id)
    return;
  // The call to hipSetDevice must come before any calls to Get, which
  // may perform initialization using the GPU.
  checkCuda(hipSetDevice(device_id));
}

__global__ void computeVmapKernel(float* depth, float* vmap, float fx_inv, float fy_inv, float cx, float cy, float depthCutoff, int height, int width)
{
  int v = threadIdx.x + blockIdx.x * blockDim.x;
  int u = threadIdx.y + blockIdx.y * blockDim.y;

  if(u < height && v < width)
  {
    float z = depth[u * width + v];

    if(z != 0 && z < depthCutoff)
    {
      float vx = z * (u - cx) * fx_inv;
      float vy = z * (v - cy) * fy_inv;
      float vz = z;

      vmap[0 + 3 * (u * width + v)] = vx;
      vmap[1 + 3 * (u * width + v)] = vy;
      vmap[2 + 3 * (u * width + v)] = vz;
    }
    else
    {
      vmap[0 + 3 * (u * width + v)] = __int_as_float(0x7fffffff); /*HIP_NAN_F*/
      vmap[1 + 3 * (u * width + v)] = __int_as_float(0x7fffffff);
      vmap[2 + 3 * (u * width + v)] = __int_as_float(0x7fffffff);
    }
  }
}

__global__ void computeNmapKernel(float* vmap, float* nmap, int height, int width)
{
  int v = threadIdx.x + blockIdx.x * blockDim.x;
  int u = threadIdx.y + blockIdx.y * blockDim.y;

  if(u >= height || v >= width)
    return;

  if(u == height - 1 || v == width - 1)
  {
    nmap[0 + 3 * (u * width + v)] = __int_as_float(0x7fffffff); /*HIP_NAN_F*/
    nmap[1 + 3 * (u * width + v)] = __int_as_float(0x7fffffff);
    nmap[2 + 3 * (u * width + v)] = __int_as_float(0x7fffffff);
    return;
  }

  Eigen::Matrix<float,3,1,Eigen::DontAlign> v00, v01, v10;
  v00(0) = vmap[0 + 3 * (u * width + v)];
  v01(0) = vmap[0 + 3 * ((u + 1) * width + v)];
  v10(0) = vmap[0 + 3 * (u * width + v + 1)];

  if(!isnan(v00(0)) && !isnan(v01(0)) && !isnan(v10(0)))
  {
    v00(1) = vmap[1 + 3 * (u * width + v)];
    v01(1) = vmap[1 + 3 * ((u + 1) * width + v)];
    v10(1) = vmap[1 + 3 * (u * width + v + 1)];

    v00(2) = vmap[2 + 3 * (u * width + v)];
    v01(2) = vmap[2 + 3 * ((u + 1) * width + v)];
    v10(2) = vmap[2 + 3 * (u * width + v + 1)];

    Eigen::Matrix<float,3,1,Eigen::DontAlign> r = (v01 - v00).cross(v10 - v00).normalized();

    nmap[0 + 3 * (u * width + v)] = r(0);
    nmap[1 + 3 * (u * width + v)] = r(1);
    nmap[2 + 3 * (u * width + v)] = r(2);
  }
  else
  {
    nmap[0 + 3 * (u * width + v)] = __int_as_float(0x7fffffff); /*HIP_NAN_F*/
    nmap[1 + 3 * (u * width + v)] = __int_as_float(0x7fffffff);
    nmap[2 + 3 * (u * width + v)] = __int_as_float(0x7fffffff);
  }
}

void compute_normals(float* depth, float* nmap, float fx, float fy, float cx, float cy, float depthCutoff, int height, int width, int device_id)
{
  set_device(device_id);

  // allocate memory
  float* depth_device;
  float* vmap_device;
  float* nmap_device;

  checkCuda(hipMalloc((void **)&depth_device, height * width * sizeof(float)));
  checkCuda(hipMalloc((void **)&vmap_device, height * width * 3 * sizeof(float)));
  checkCuda(hipMalloc((void **)&nmap_device, height * width * 3 * sizeof(float)));
  checkCuda(hipMemcpy(depth_device, depth, height * width * sizeof(float), hipMemcpyHostToDevice));

  // compute vmap
  dim3 block(32, 8);
  dim3 grid(1, 1, 1);
  grid.x = divUp(width, block.x);
  grid.y = divUp(height, block.y);

  computeVmapKernel<<<grid, block>>>(depth_device, vmap_device, 1.f / fx, 1.f / fy, cx, cy, depthCutoff, height, width);
  checkCuda(hipGetLastError());

  // compute nmap
  computeNmapKernel<<<grid, block>>>(vmap_device, nmap_device, height, width);
  checkCuda(hipGetLastError());

  // copy output
  checkCuda(hipMemcpy(nmap, nmap_device, height * width * 3 * sizeof(float), hipMemcpyDeviceToHost));

  // clean up
  checkCuda(hipFree(depth_device));
  checkCuda(hipFree(vmap_device));
  checkCuda(hipFree(nmap_device));
}
